#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <math.h>

// Thread block size
#define BLOCK_SIZE 16

// Matrix dimensions
// (chosen as multiples of the thread block size for simplicity)
#define WA (3 * BLOCK_SIZE) // Matrix A width
#define HA (5 * BLOCK_SIZE) // Matrix A height
#define WB (8 * BLOCK_SIZE) // Matrix B width
#define HB WA  // Matrix B height
#define WC WB  // Matrix C width 
#define HC HA  // Matrix C height


#define CHECK_BANK_CONFLICTS 0
#if CHECK_BANK_CONFLICTS
#define AS(i, j) cutilBankChecker(((float*)&As[0][0]), (BLOCK_SIZE * i + j))
#define BS(i, j) cutilBankChecker(((float*)&Bs[0][0]), (BLOCK_SIZE * i + j))
#else
#define AS(i, j) As[i][j]
#define BS(i, j) Bs[i][j]
#endif

////////////////////////////////////////////////////////////////////////////////
//! Matrix multiplication on the device: C = A * B
//! wA is A's width and wB is B's width
////////////////////////////////////////////////////////////////////////////////
__global__ void
matrixMul( float* C, float* A, float* B, int wA, int wB)
{
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Index of the first sub-matrix of A processed by the block
    int aBegin = wA * BLOCK_SIZE * by;

    // Index of the last sub-matrix of A processed by the block
    int aEnd   = aBegin + wA - 1;

    // Step size used to iterate through the sub-matrices of A
    int aStep  = BLOCK_SIZE;

    // Index of the first sub-matrix of B processed by the block
    int bBegin = BLOCK_SIZE * bx;

    // Step size used to iterate through the sub-matrices of B
    int bStep  = BLOCK_SIZE * wB;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    float Csub = 0;

    // Loop over all the sub-matrices of A and B
    // required to compute the block sub-matrix
    for (int a = aBegin, b = bBegin;
             a <= aEnd;
             a += aStep, b += bStep) {

        // Declaration of the shared memory array As used to
        // store the sub-matrix of A
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

        // Declaration of the shared memory array Bs used to
        // store the sub-matrix of B
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        // Load the matrices from device memory
        // to shared memory; each thread loads
        // one element of each matrix
        AS(ty, tx) = A[a + wA * ty + tx];
        BS(ty, tx) = B[b + wB * ty + tx];

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
        for (int k = 0; k < BLOCK_SIZE; ++k)
            Csub += AS(ty, k) * BS(k, tx);

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write the block sub-matrix to device memory;
    // each thread writes one element
    int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    C[c + wB * ty + tx] = Csub;
}

void randomInit(float* data, int size)
{
    for (int i = 0; i < size; ++i)
        data[i] = rand() / (float)RAND_MAX;
}

void
runTest(int argc, char** argv)
{
    // set seed for rand()
    srand(2006);

    // allocate host memory for matrices A and B
    unsigned int size_A = WA * HA;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float* h_A = (float*) malloc(mem_size_A);
    unsigned int size_B = WB * HB;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float* h_B = (float*) malloc(mem_size_B);

    // initialize host memory
    randomInit(h_A, size_A);
    randomInit(h_B, size_B);

    // allocate device memory
    float* d_A;
    hipMalloc((void**) &d_A, mem_size_A);
    float* d_B;
    hipMalloc((void**) &d_B, mem_size_B);

    // copy host memory to device
    hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice);

    // allocate device memory for result
    unsigned int size_C = WC * HC;
    unsigned int mem_size_C = sizeof(float) * size_C;
    float* d_C;
    hipMalloc((void**) &d_C, mem_size_C);

    // allocate host memory for the result
    float* h_C = (float*) malloc(mem_size_C);
    
    // setup execution parameters
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(WC / threads.x, HC / threads.y);

    // execute the kernel
    matrixMul<<< grid, threads >>>(d_C, d_A, d_B, WA, WB);

    // copy result from device to host
    hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost);

    // clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    hipDeviceReset();
}
