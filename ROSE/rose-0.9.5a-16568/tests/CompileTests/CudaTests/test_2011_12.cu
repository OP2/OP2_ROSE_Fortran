
#include <hip/hip_runtime.h>

__global__ void square_array() {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
}

int main(int argc, char ** argv) {
	dim3 threads(1, 1);
	dim3 grid(1, 1, 1);
	
	square_array <<< threads, grid >>> ();
}
